
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define N (1024*1024)
#define M (10000)

float data[N];
__global__ void cudakernel(float *d_data)
{
   int i = threadIdx.x + blockIdx.x * blockDim.x;
   //d_data[i] = 1.0f * i / N;
   d_data[i] *= d_data[i];
   for(int j = 0; j < M; j++)
      d_data[i]++;
}

void graphic()
{
  float *d_data;
 
  hipMalloc(&d_data, N * sizeof(float));
  for(int j = 0; j < N; j++)
      data[j] = j;
  
  hipMemcpy(d_data, data, N * sizeof(float), hipMemcpyHostToDevice);
  
  cudakernel<<<N/256, 256>>>(d_data);
  
  hipMemcpy(data, d_data, N * sizeof(float), hipMemcpyDeviceToHost);
  
  hipFree(d_data); 
}
int main()
{

  graphic();
  int sel = 253454;

   //printf("Enter an index: ");
    //scanf("%d", &sel);
   printf("data[%d] = %f\n", sel, data[sel]);
}