#include <thrust/version.h>
#include <iostream>

int main(void)
{
  int major = THRUST_MAJOR_VERSION;
  int minor = THRUST_MINOR_VERSION;

  std::cout << "Thrust v" << major << "." << minor << std::endl;

  return 0;
}

//https://solarianprogrammer.com/2012/05/31/matrix-multiplication-cuda-cublas-hiprand-thrust/
//http://stackoverflow.com/questions/7663343/simplest-possible-example-to-show-gpu-outperform-cpu-using-cuda
//https://www.google.co.in/webhp?sourceid=chrome-instant&ion=1&espv=2&ie=UTF-8#q=cuda%20libraries